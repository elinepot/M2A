/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

int main (void){

	/*************************************************************

	Once requested, replace this comment by the appropriate code

	*************************************************************/

	return 0;
}